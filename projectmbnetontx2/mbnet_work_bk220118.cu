#include "hip/hip_runtime.h"
#include "slenet_params.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/param.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "mnist.h"
#include "cublas_utils.h"
//#include "utils_user.h"

#include "smbnet.h"
//#include "mblenet5.h"

#define DBG 1

int print_status = 1; 


// Defining the Layer class
class Layer {
	public:
		int M, N, O;
		float *pre_output, *output;
		float *weight, *bias;
    float *im2col_A; //for im2col
    float *gemm_B;
    float *gemm_C;

		Layer(int M, int N, int O);
		~Layer();

    void clear();
};

Layer::Layer(int M, int N, int O) {
	this->M = M;
	this->N = N;
	this->O = O;

	float *temp_weight, *temp_bias;

	// Initializing weights and biases
	temp_weight = (float*)malloc(sizeof(float) * M * N);
	temp_bias = (float*)malloc(sizeof(float) * N);

  temp_weight[0] = 0.0f; 
	for (int i = 0; i < M * N; i++){
		temp_weight[i] = WEIGHT; //1.0f;
    //temp_weight[i+1] = temp_weight[i]+1.0f; 
  }

	for (int i = 0; i < N; i++)
		temp_bias[i] = BIAS; //1.0f;

	// Allocating space for CUDA variables
	hipMalloc(&pre_output, sizeof(float) * O);
	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&weight, sizeof(float) * M * N);
	hipMalloc(&bias, sizeof(float) * N);

  hipMalloc(&im2col_A, sizeof(float) *M*O/N);
  hipMalloc(&gemm_B, sizeof(float) *M*N);
  hipMalloc(&gemm_C, sizeof(float) *(O/N)*N);

	// Copying weights and biases to CUDA variables
	hipMemcpy(weight, temp_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
	hipMemcpy(bias, temp_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	// Freeing temporary weights and biases
	free(temp_weight);
	free(temp_bias);
}

Layer::~Layer() {
	// Freeing all CUDA varibles of a layer
	hipFree(pre_output);
	hipFree(output);
	hipFree(weight);
	hipFree(bias);
  hipFree(im2col_A);
}

void Layer::clear(){
	hipMemset(pre_output, 0x00, sizeof(float)*O); 
	hipMemset(output, 0x00, sizeof(float)*O); 
}

// Initializing a convolutional layer
Layer conv_layer(FILTER_SIZE * FILTER_SIZE, CHANNEL, CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
Layer ss_layer(SS_SIZE * SS_SIZE, SS_CHANNELS, CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
Layer fc_layer(CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, NUM_CLASSES, NUM_CLASSES);

#if LENET5_C2
Layer conv2_layer(FILTER_SIZE2 * FILTER_SIZE2, CHANNEL2, CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE);
#endif 
#if LENET5_SS2
Layer ss2_layer(SS2_SIZE * SS2_SIZE, SS2_CHANNELS, CHANNEL2 * SS2_OUTPUT_SIZE * SS2_OUTPUT_SIZE);
#endif 

double time_taken = 0.0;

#if CONV_COMPOSITE
__global__ void kernel_conv1_composite(float input[INCH][INSIZE][INSIZE], 
                                      float pre_output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], 
                                      float weight[CHANNEL][FILTER_SIZE][FILTER_SIZE],
                                      float bias[CHANNEL], 
                                      float output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE]) {

  #if CONV_SHARED
	int tidx = threadIdx.x;
	int bIdx = blockIdx.x;
	
	__shared__ float sh_img[SHBS][SHBS];
	__shared__ float sh_weight[CHANNEL][FILTER_SIZE][FILTER_SIZE];

	#if 1
	int img_row = tidx / SHBS; 
	int img_col = tidx % SHBS; 
	#else 
	int img_row = (itemp/=1) % SHBS;
	int img_col = (itemp/=SHBS) % SHBS;
	#endif 

	#if 1
	int bIdx_r = bIdx / GRID; 
	int bIdx_c = bIdx % GRID; 
	#else 
	int itemp = bIdx; 
	int bIdx_r = (itemp/=1) % GRID;
	int bIdx_c = (itemp/=GRID) % GRID;
	#endif 

	/* input image copy to shared memory */ 
	if (tidx < SHBS*SHBS){
		//sh_img[img_row][img_col] = 0; 
		sh_img[img_row][img_col] = input[blockIdx.y][bIdx_r*BS + img_row][bIdx_c*BS + img_col];
	}
  	__syncthreads();
	

	#if 1
	int ch = tidx / (FILTER_SIZE*FILTER_SIZE);
	int k_row = (tidx % (FILTER_SIZE*FILTER_SIZE)) / FILTER_SIZE;
	int k_col = (tidx % (FILTER_SIZE*FILTER_SIZE)) % FILTER_SIZE;
	#else 
	itemp = tidx;
	int ftr = (itemp/=1)%CHANNEL; 
	int k_row = (itemp/=CHANNEL) % FILTER_SIZE;
	int k_col = (itemp/=FILTER_SIZE) % FILTER_SIZE;
	#endif 
	
	/* kernel filter copy to shared memory */ 
	if (tidx < CHANNEL*FILTER_SIZE*FILTER_SIZE){
		//sh_weight[ch][k_row][k_col] = 0; 
		sh_weight[ch][k_row][k_col] = weight[ch][k_row][k_col];
	}
	__syncthreads();

	ch = tidx/(BS*BS);
	int w_row = (tidx % (BS*BS)) / BS;
	int w_col = (tidx % (BS*BS)) % BS; 

	float sum = 0;
	if (w_row < BS && w_col < BS && ch < CHANNEL) {
		for (int i = 0; i < FILTER_SIZE; i++)
			for (int j = 0; j < FILTER_SIZE; j++)
				sum += sh_img[w_row + i][w_col + j] * sh_weight[ch][i][j];
		pre_output[ch][bIdx_r * BS + w_row][bIdx_c * BS + w_col] = sum;
    	pre_output[ch][bIdx_r * BS + w_row][bIdx_c * BS + w_col] += bias[ch];
    	output[ch][bIdx_r * BS + w_row][bIdx_c * BS + w_col] = 1 / (1 + exp(-pre_output[ch][bIdx_r * BS + w_row][bIdx_c *  BS + w_col]));    
  }

  #else
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
	float tempC = 0.0f;

	for (int i = 0; i < FILTER_SIZE; i++) {
		for (int j = 0; j < FILTER_SIZE; j++) {
			tempC += weight[channel][i][j] * input[i + output_x][j + output_y];
		}
	}
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE){
	    pre_output[channel][output_x][output_y] = tempC;
      pre_output[channel][output_x][output_y] += bias[channel];
      output[channel][output_x][output_y] = 1 / (1 + exp(-pre_output[channel][output_x][output_y]));
    }

  #endif 
}
#else

__global__ void kernel_conv_filter(float input[INCH][INSIZE][INSIZE], 
                                  float pre_output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], 
                                  float weight[CHANNEL][FILTER_SIZE][FILTER_SIZE]) {
	#if CONV_SHARED
	int tidx = threadIdx.x;
	int bIdx = blockIdx.x;
	
	__shared__ float sh_img[SHBS][SHBS];
	__shared__ float sh_weight[CHANNEL][FILTER_SIZE][FILTER_SIZE];

	#if 1
	int img_row = tidx / SHBS; 
	int img_col = tidx % SHBS; 
	#else 
	int img_row = (itemp/=1) % SHBS;
	int img_col = (itemp/=SHBS) % SHBS;
	#endif 

	#if 1
	int bIdx_r = bIdx / GRID; 
	int bIdx_c = bIdx % GRID; 
	#else 
	int itemp = bIdx; 
	int bIdx_r = (itemp/=1) % GRID;
	int bIdx_c = (itemp/=GRID) % GRID;
	#endif 

	/* input image copy to shared memory */ 
	if (tidx < SHBS*SHBS){
		//sh_img[img_row][img_col] = 0; 
		sh_img[img_row][img_col] = input[blockIdx.y][bIdx_r*BS + img_row][bIdx_c*BS + img_col];
	}
  	__syncthreads();
	

	#if 1
	int ch = tidx / (FILTER_SIZE*FILTER_SIZE);
	int k_row = (tidx % (FILTER_SIZE*FILTER_SIZE)) / FILTER_SIZE;
	int k_col = (tidx % (FILTER_SIZE*FILTER_SIZE)) % FILTER_SIZE;
	#else 
	itemp = tidx;
	int ftr = (itemp/=1)%CHANNEL; 
	int k_row = (itemp/=CHANNEL) % FILTER_SIZE;
	int k_col = (itemp/=FILTER_SIZE) % FILTER_SIZE;
	#endif 
	
	/* kernel filter copy to shared memory */ 
	if (tidx < CHANNEL*FILTER_SIZE*FILTER_SIZE){
		//sh_weight[ch][k_row][k_col] = 0; 
		sh_weight[ch][k_row][k_col] = weight[ch][k_row][k_col];
	}
	__syncthreads();

	ch = tidx/(BS*BS);
	int w_row = (tidx % (BS*BS)) / BS;
	int w_col = (tidx % (BS*BS)) % BS; 

	float sum = 0;
	if (w_row < BS && w_col < BS && ch < CHANNEL) {
		for (int i = 0; i < FILTER_SIZE; i++)
			for (int j = 0; j < FILTER_SIZE; j++)
				sum += sh_img[w_row + i][w_col + j] * sh_weight[ch][i][j];
		pre_output[ch][bIdx_r * BS + w_row][bIdx_c * BS + w_col] = sum;
	}	

	#else
  	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
	
	float tempC = 0.0f;
	for (int i = 0; i < FILTER_SIZE; i++) {
		for (int j = 0; j < FILTER_SIZE; j++) {
			tempC += weight[channel][i][j] * input[i + output_x][j + output_y];
		}
	}
  	if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
		pre_output[channel][output_x][output_y] = tempC;
	#endif 
}
__global__ void kernel_conv_bias(float pre_output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float bias[CHANNEL]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] += bias[channel];
}

__global__ void kernel_conv_sigmoid(float preact[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], 
                          float output[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  output[channel][output_x][output_y] = 1 / (1 + exp(-preact[channel][output_x][output_y]));
}
#endif 

#if SS_COMPOSITE
__global__ void kernel_ss1_composite(float input[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE],
#else
__global__ void kernel_ss1_filter(float input[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE],
#endif  
								float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], 
								float weight[SS_CHANNELS][SS_SIZE][SS_SIZE],
								float bias[SS_CHANNELS], float output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
	float tempC = 0.0f;

	for (int i = 0; i < SS_SIZE; i++) {
		for (int j = 0; j < SS_SIZE; j++) {
			tempC += weight[0][i][j] * input[channel][i + output_x * SS_STRIDE][j + output_y * SS_STRIDE];
		}
	}
  	if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE){
		pre_output[channel][output_x][output_y] = tempC;
		#if SS_COMPOSITE
		pre_output[channel][output_x][output_y] += bias[0];
		output[channel][output_x][output_y] = 1 / (1 + exp(-pre_output[channel][output_x][output_y]));
		#endif
	}
}

__global__ void kernel_ss1_bias(float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float bias[SS_CHANNELS]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
  if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE)
	  pre_output[channel][output_x][output_y] += bias[0];
}

__global__ void kernel_ss1_sigmoid(float pre_output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], float output[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % CHANNEL;
	int output_x = (idx / CHANNEL) % SS_OUTPUT_SIZE;
	int output_y = (idx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;
  if (idx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE)
	  output[channel][output_x][output_y] = 1 / (1 + exp(-pre_output[channel][output_x][output_y]));
}

#if FC_COMPOSITE
__global__ void kernel_fc1_composite(float input[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], 
#else 
__global__ void kernel_fc1(float input[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE], 
#endif 
							float pre_output[NUM_CLASSES], 
                            float weight[NUM_CLASSES][CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE],
							float bias[NUM_CLASSES], float output[NUM_CLASSES]) {

  	#if FC_SHARED
  	//int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidx = threadIdx.x; 	  
	int bidx = blockIdx.x; 
  	__shared__ float tempA[NUM_CLASSES][CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE]; 

  	int ch = tidx  % CHANNEL;
  	int x = (tidx /  CHANNEL) % SS_OUTPUT_SIZE;
  	int y = (tidx / CHANNEL / SS_OUTPUT_SIZE) % SS_OUTPUT_SIZE;

  	float tempC = 0.0f; 
  	if (tidx < CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE){
		//input[ch][x][y] = 1.0f; 
  		tempA[blockIdx.x][ch][x][y] =  weight[blockIdx.x][ch][x][y] * input[ch][x][y];
		__syncthreads(); 
	}

	if (tidx  == (blockDim.x-1)){
		for(int i=0; i<CHANNEL ; i++)
			for(int j=0; j<SS_OUTPUT_SIZE ; j++)
				for(int k=0; k<SS_OUTPUT_SIZE ; k++)
					tempC += tempA[blockIdx.x][i][j][k]; 
		pre_output[blockIdx.x] = tempC;
		#if FC_COMPOSITE
		pre_output[bidx] += bias[bidx];
		output[bidx] = 1 / (1 + exp(-pre_output[bidx]));
		#endif 
  	}

 	#else
  	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	float tempC = 0.0f;

  	if (idx < NUM_CLASSES){
		for (int i = 0; i < CHANNEL; i++) {
			for (int j = 0; j < SS_OUTPUT_SIZE; j++) {
				for (int k = 0; k < SS_OUTPUT_SIZE; k++) {
					tempC += weight[channel][i][j][k] * input[i][j][k];
				}
			}
		}
		pre_output[channel] = tempC;
		#if FC_COMPOSITE
		pre_output[channel] += bias[channel];
		output[channel] = 1 / (1 + exp(-pre_output[channel]));
		#endif 
	}
  #endif                           
}

__global__ void kernel_fc1_bias(float pre_output[NUM_CLASSES], float bias[NUM_CLASSES]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	pre_output[channel] += bias[channel];
}

__global__ void kernel_fc1_sigmoid(float pre_output[NUM_CLASSES], float output[NUM_CLASSES]) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int channel = idx % NUM_CLASSES;
	output[channel] = 1 / (1 + exp(-pre_output[channel]));
}

//%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
__global__ void ker2row_kernel(float weight_col[CHANNEL][FILTER_SIZE*FILTER_SIZE], float weight[CHANNEL][FILTER_SIZE][FILTER_SIZE]) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  
  int channel = idx % CHANNEL;
	int x = (idx / CHANNEL) % FILTER_SIZE;
	int y = (idx / CHANNEL / FILTER_SIZE) % FILTER_SIZE;
  if (idx < CHANNEL*FILTER_SIZE*FILTER_SIZE)
	  weight_col[channel][x*FILTER_SIZE + y] = weight[channel][x][y];
}

__global__ void gemm_global_kernel(float matB[CHANNEL][FILTER_SIZE*FILTER_SIZE] 
                      ,float matA[FILTER_SIZE*FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE]
                      ,float matC[CHANNEL][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE]) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x; 

  int x = idx % CHANNEL;
	int y = (idx / CHANNEL) % (CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);

  float tempC; 
  //matC[x][y] = 0.0f; 
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE){
    for(int i=0; i<FILTER_SIZE*FILTER_SIZE; i++){
      tempC += matB[x][i] * matA[i][y]; 
    } 
    matC[x][y] = tempC; 
  }
}

__global__ void col2im_kernel(float preout[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE], float preout_col[CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE][CHANNEL]) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  
  int channel = idx % CHANNEL;
	int x = (idx / CHANNEL) % CONV_OUTPUT_SIZE;
	int y = (idx / CHANNEL / CONV_OUTPUT_SIZE) % CONV_OUTPUT_SIZE;
  if (idx < CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE)
	  preout[channel][x][y] = preout_col[x*FILTER_SIZE + y][channel];
}

///*
void verifyConv(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE);
}

#if LENET5_C2
void verifyConv2(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
		//printf("%.1f ", maxError); 
    if (maxError > 0.2)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL2*CONV2_OUTPUT_SIZE*CONV2_OUTPUT_SIZE);
}
#endif 


#if 1
void debugPrintConv(float A[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE]){
  //if (print_status == 1){
  
      for(int i=0; i<CHANNEL; i++){
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.2f ", A[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
   // print_status--; 
    printf("\n\n");
  //}
}
#endif 

void verify_im2col(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, FILTER_SIZE * FILTER_SIZE *CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE);
}


void verify_ker2row(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0; 
	for (int i = 0; i < CHANNEL * FILTER_SIZE * FILTER_SIZE; i++){ 
		maxError = max(abs(A[i] - val), maxError);
    if (maxError != 0)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*FILTER_SIZE*FILTER_SIZE);
}

void verifySS(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE; i++){
		maxError = max(abs(A[i] - val), maxError);
    	if (maxError > 0.007f)
      		cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE);
}

#if LENET5_SS2
void verifySS2(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < CHANNEL2 * SS2_OUTPUT_SIZE * SS2_OUTPUT_SIZE; i++){
		maxError = max(abs(A[i] - val), maxError);
    	if (maxError > 0.007f)
      		cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, CHANNEL2*SS2_OUTPUT_SIZE*SS2_OUTPUT_SIZE);
}
#endif 

void verifyFCfil(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < NUM_CLASSES; i++){
		maxError = max(abs(A[i] - val), maxError);
    	if (maxError > 4.0f)
      		cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, NUM_CLASSES);
}


void verifyFC(float *A, float val) {
	float maxError = 0.0f;

  int cnt = 0;
	for (int i = 0; i < NUM_CLASSES; i++){
		maxError = max(abs(A[i] - val), maxError);
    if (maxError > 0.0009f)
      cnt++; 
  }
	printf("maxError = %f (cnt = %d),%d)\n", maxError, cnt, NUM_CLASSES);
}
//*/
// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i, n) \
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
        i < (n); \
        i += blockDim.x * gridDim.x)

// https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {

    int index = blockIdx.x*blockDim.x+threadIdx.x;

    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //data_im[(channel_in * height + h_in) * width + w_in + i * width + j];
                //(*data_col_ptr) = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}


// Performing a forward pass using a single image
static double forward_pass(double data[INSIZE][INSIZE], bool verify) {
	// Copying a double data to a float data
	float input[INSIZE][INSIZE];
	float *verification;
  
  input[0][0] = 0.0f; 
	for (int i = 0; i < INSIZE; i++) {
		for (int j = 0; j < INSIZE; j++){
			#if SIMULATION
      		input[i][j] = INPUT; //Simulated data
			#else
			input[i][j] = data[i][j];  //MNIST data
			#endif 
      //printf("%f ",input[i][j]);
    }
    //printf("\n");
	}

//Layer conv_layer(FILTER_SIZE * FILTER_SIZE, CHANNEL, CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
//Layer ss_layer(SS_SIZE * SS_SIZE, SS_CHANNELS, CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
//Layer fc_layer(CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, NUM_CLASSES, NUM_CLASSES);

  //l_input.clear(); 
	//conv_layer.clear(); //need to be checked !!!
	//ss_layer.clear();
	//fc_layer.clear();

	float (*d_input)[INSIZE][INSIZE];
	hipMalloc(&d_input, sizeof(float) * INSIZE * INSIZE * INCH);
	hipMemcpy(d_input, input, sizeof(float) * INSIZE * INSIZE * INCH, hipMemcpyHostToDevice);

  //For im2col workspace 
  //float* im2col_input;  
  //hipMalloc(&im2col_input, sizeof(float) * INSIZE * INSIZE);

  //float* im2col_workspace;  
  //hipMalloc(&im2col_workspace, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

#if CPU_GEMM
float matB[CHANNEL][FILTER_SIZE * FILTER_SIZE];
float matA[FILTER_SIZE * FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE];
float matC[CHANNEL][CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE];
#endif 
//float newMatC[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE];
//float gemmMatC[CHANNEL][CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE];

#if CONV_COMPOSITE
	// Performing Convolutional composite
	#if CONV_SHARED
	const dim3 numBlocks(CONV_NB,1);
  	const dim3 threadsPerBlock(CONV_TPB);
	kernel_conv1_composite<<<numBlocks, threadsPerBlock>>>(d_input, 
	#else
	kernel_conv1_composite<<<(N1+K1-1)/K1, K1>>>(d_input, 
	#endif  
                                            (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output,
                                            (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight,
                                            conv_layer.bias, 
                                            (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output);

	// Verifying Convolutional composite
	if (verify) {
		printf("Veri Convolutional composite: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE + BIAS); //25.0f
		free(verification);
	}

#elif DIRECT 
	// Performing Convolutional filtering
  	#if CONV_SHARED
	const dim3 numBlocks(CONV_NB,1);
  	const dim3 threadsPerBlock(CONV_TPB);
	kernel_conv_filter<<<numBlocks, threadsPerBlock>>>(d_input, 
	#else
	kernel_conv_filter<<<(N1+K1-1)/K1, K1>>>(d_input,
	#endif  
                                            (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output,
                                            (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight);

	// Verifying Convolutional filtering operation
	if (verify) {
		printf("Veri Convolutional filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
	}

#else // GEMM // gemm or direct setting 
  //im2col_gpu_kernel_ext<<<(N1+K1-1)/K1, K1>>>(CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE, d_input, INSIZE, INSIZE, FILTER_SIZE, FILTER_SIZE, 0, 0, STRIDE, STRIDE, 1, 1, CONV_OUTPUT_SIZE, CONV_OUTPUT_SIZE,ic_workspace);
///*
  im2col_gpu_kernel<<<(N11+K11-1)/K11, K11>>>(CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE, //num_kernels, = channels * height_col * width_col; 
                                              (float *)d_input, //data_im, 
                                              INSIZE, //height, 
                                              INSIZE, //width, 
                                              FILTER_SIZE, //ksize, 
                                              0, //pad,
                                              STRIDE, //stride, 
                                              CONV_OUTPUT_SIZE, //height_col, 
                                              CONV_OUTPUT_SIZE, //width_col, 
                                              (float *)conv_layer.im2col_A); //data_col);

  	// Verifying im2col operation
	if (verify) { //verify
    if (print_status == 1){
		printf("Verifying im2col_A: ");
		verification = (float*)malloc(sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.im2col_A, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verify_im2col(verification, INPUT); //-1.0f 

    #if 0
    for(int i=0; i<INSIZE*INSIZE; i++){ 
      
      if (i%(INSIZE) == 0){
           printf("\n");
      }
      printf("%2.1f ", verification[i]);
    }
    printf("\n");
    #endif 
    free(verification);
    print_status--;
    }
  }                                     
		
  //ker2col operation 
  ker2row_kernel<<<CHANNEL, FILTER_SIZE * FILTER_SIZE>>>((float(*)[FILTER_SIZE*FILTER_SIZE])conv_layer.gemm_B, 
                                                    (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight); 
// Verifying ker2row operation
	if (verify) { //verify
		printf("Verifying ker2row_A: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * FILTER_SIZE * FILTER_SIZE);
		hipMemcpy(verification, conv_layer.gemm_B, sizeof(float) * CHANNEL * FILTER_SIZE * FILTER_SIZE, hipMemcpyDeviceToHost);
		verify_ker2row(verification, WEIGHT); //-1.0f 
		free(verification); 
		print_status--; 
	}

///* //on cpu gemm 	
#if CPU_GEMM 
  //float matA[FILTER_SIZE * FILTER_SIZE][CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE];
  hipMemcpy(matA, conv_layer.im2col_A, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  //float matB[CHANNEL][FILTER_SIZE * FILTER_SIZE]; 
  hipMemcpy(matB, conv_layer.gemm_B, sizeof(float) * FILTER_SIZE * FILTER_SIZE * CHANNEL, hipMemcpyDeviceToHost);
 
  #if 1
  //float matC[CHANNEL][CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE]; 
  //gemm_custom_cpu(); 
  {

    for(int i=0; i<CHANNEL ; i++){
      for(int j=0; j< CONV_OUTPUT_SIZE*CONV_OUTPUT_SIZE ; j++){
        matC[i][j] = 0.0f;
        for(int k=0; k< FILTER_SIZE*FILTER_SIZE; k++){
          matC[i][j] += matB[i][k] * matA[k][j]; 
        }
      }
    }
  }
  #endif 

	//kernel_conv_filter<<<(N1+K1-1)/K1, K1>>>(d_input, 
  //                                          (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output,
  //                                          (float(*)[FILTER_SIZE][FILTER_SIZE])conv_layer.weight);

  hipMemcpy(conv_layer.pre_output, matC, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyHostToDevice);
  //hipMemcpy(conv_layer.pre_output, newMatC, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyHostToDevice); //both are okay

#elif GEMM_GLOBAL// GPU_GEMM 

  gemm_global_kernel<<<CHANNEL, CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE>>>((float(*)[FILTER_SIZE*FILTER_SIZE])conv_layer.gemm_B 
                      ,(float(*)[CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE])conv_layer.im2col_A
                      //,(float(*)[CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE])conv_layer.gemm_C); 
                      ,(float(*)[CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE])conv_layer.pre_output);


#else  //using hipblasSgemm

  int m = CHANNEL; // l.n / l.groups
  int k = FILTER_SIZE * FILTER_SIZE; // l.size*l.size
  int n = CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE; //l.out_w*l.out_h

  float *a = conv_layer.gemm_B; //l.weights_gpu + j*l.nweights / l.groups;
  float *b = conv_layer.im2col_A; //state.workspace
  float *c = conv_layer.pre_output; //l.output_gpu + (i*l.groups + j)*n*m;

  //gemm_ongpu(0, 0, m, n, k, 1, a, k, b, n, 1, c, n); 
  const float alpha = 1, beta = 0;
  hipblasHandle_t handle = blas_handle();
  hipError_t status = (hipError_t)hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                         n, m, k, &alpha, b, n, a, k, &beta, c, n);
  //hipblasDestroy(handle);

	// Verifying hipblasSgemm operation
	if (verify) {
		printf("Veri hipblasSgemm: ");
		verification = (float*)malloc(sizeof(float) * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE * CHANNEL);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE * CHANNEL, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE); //25.0f
		free(verification);
	}

  #if 0
  if (print_status == 1){
    //verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
    hipMemcpy(newMatC, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
  
      for(int i=0; i<1; i++){ //CHANNEL
        for(int j=0; j<CONV_OUTPUT_SIZE; j++){
          for(int k=0; k<CONV_OUTPUT_SIZE ; k++){
            printf("%3.1f ", newMatC[i][j][k]);
          }
          printf("\n");
        }   
        printf("\n");
    }
    print_status--; 
    printf("\n\n");
  }
  #endif 

#endif //hipblasSgemm 
#endif //GEMM


#if !CONV_COMPOSITE
	// Performing Convolutional bias addition
	kernel_conv_bias<<<(N1+K1-1)/K1, K1>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output, conv_layer.bias);

	// Verifying Convolutional bias operation
	if (verify) {
		printf("Veri Convolutional bias: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.pre_output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, INPUT*WEIGHT*FILTER_SIZE*FILTER_SIZE + BIAS); // 26.0f
		free(verification);
	}

	// Performing Convolutional sigmoid operation
	kernel_conv_sigmoid<<<(N1+K1-1)/K1, K1>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.pre_output, (float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output);

	// Verifying Convolutional sigmoid operation
	if (verify) {
		printf("Veri Convolutional sigmoid: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE);
		hipMemcpy(verification, conv_layer.output, sizeof(float) * CHANNEL * CONV_OUTPUT_SIZE * CONV_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv(verification, CONV_POST_ACT); // 1.0f
		free(verification);
	}
#endif //!CONV_COMPOSITE

#if ALL_LAYERS 

	// Performing Subsampling filtering
	#if SS_COMPOSITE 
	kernel_ss1_composite<<<(N2+K2-1)/K2, K2>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output, 
	#else
	kernel_ss1_filter<<<(N2+K2-1)/K2, K2>>>((float(*)[CONV_OUTPUT_SIZE][CONV_OUTPUT_SIZE])conv_layer.output, 
	#endif 
											(float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, 
											(float(*)[SS_SIZE][SS_SIZE])ss_layer.weight, 
											ss_layer.bias, (float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output);

	// Verifying Subsampling filtering operation
	if (verify) {
		printf("Veri Subsampling filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		#if SS_COMPOSITE
		hipMemcpy(verification, ss_layer.output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
        verifySS(verification, SS_POST_ACT); //1.0f
		#else
		hipMemcpy(verification, ss_layer.pre_output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, CONV_POST_ACT*WEIGHT*SS_SIZE*SS_SIZE); //16.0f
		#endif 
		free(verification);
	}

	#if !SS_COMPOSITE
	// Performing Subsampling bias addition
	kernel_ss1_bias<<<(N2+K2-1)/K2, K2>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, ss_layer.bias);

	// Verifying Subsampling bias operation
	if (verify) {
		printf("Veri Subsampling bias: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		hipMemcpy(verification, ss_layer.pre_output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, CONV_POST_ACT*WEIGHT*SS_SIZE*SS_SIZE + BIAS); // 17.0f
		free(verification);
	}

	// // Performing Subsampling sigmoid operation
	kernel_ss1_sigmoid<<<(N2+K2-1)/K2, K2>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.pre_output, (float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output);

	// Verifying Subsampling sigmoid operation
	if (verify) {
		printf("Veri Subsampling sigmoid: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE);
		hipMemcpy(verification, ss_layer.output, sizeof(float) * CHANNEL * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS(verification, SS_POST_ACT); //1.0f
		free(verification);
	}
	#endif 

	//Layer conv2_layer(FILTER_SIZE2 * FILTER_SIZE2, CHANNEL2, CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE);
	#if LENET5_C2
	const dim3 numBlocks2(CONV_NB2,INNR);
  	const dim3 threadsPerBlock2(CONV_TPB2);
	kernel_conv2_filter<<<numBlocks2, threadsPerBlock2>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output, //d_input, 
  	
	//kernel_conv2_filter<<<(N1+K1-1)/K1, K1>>>(d_input,
	 
                                            (float(*)[CONV2_OUTPUT_SIZE][CONV2_OUTPUT_SIZE])conv2_layer.pre_output,
                                            (float(*)[FILTER_SIZE2][FILTER_SIZE2])conv2_layer.weight,
											conv2_layer.bias, (float(*)[CONV2_OUTPUT_SIZE][CONV2_OUTPUT_SIZE])conv2_layer.output);			


	// Verifying Convolutional filtering operation
	if (verify) {
		printf("Veri Convolutional 2 filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE);
		hipMemcpy(verification, conv2_layer.pre_output, sizeof(float) * CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv2(verification, SS_POST_ACT*WEIGHT*FILTER_SIZE2*FILTER_SIZE2 + BIAS); // 24.8  25.0f
		
		printf("Veri Convolutional 2 activation: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE);
		hipMemcpy(verification, conv2_layer.output, sizeof(float) * CHANNEL2 * CONV2_OUTPUT_SIZE * CONV2_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifyConv2(verification, CONV2_POST_ACT); // 1.0f 
		
		free(verification);
	}
	#endif 

	#if LENET5_SS2
	kernel_ss2_composite<<<(N22+K22-1)/K22, K22>>>((float(*)[CONV2_OUTPUT_SIZE][CONV2_OUTPUT_SIZE])conv2_layer.output, 
											(float(*)[SS2_OUTPUT_SIZE][SS2_OUTPUT_SIZE])ss2_layer.pre_output, 
											(float(*)[SS2_SIZE][SS2_SIZE])ss2_layer.weight, 
											ss_layer.bias, (float(*)[SS2_OUTPUT_SIZE][SS2_OUTPUT_SIZE])ss2_layer.output);

	// Verifying Subsampling filtering operation
	if (verify) {
		printf("Veri Subsampling2 filtering: ");
		verification = (float*)malloc(sizeof(float) * CHANNEL2 * SS2_OUTPUT_SIZE * SS2_OUTPUT_SIZE);
		hipMemcpy(verification, ss2_layer.pre_output, sizeof(float) * CHANNEL2 * SS2_OUTPUT_SIZE * SS2_OUTPUT_SIZE, hipMemcpyDeviceToHost);
		verifySS2(verification, CONV2_POST_ACT*WEIGHT*SS2_SIZE*SS2_SIZE + BIAS); //16.0f
	
		printf("Veri Subsampling2 activation: ");
		hipMemcpy(verification, ss2_layer.output, sizeof(float) * CHANNEL2 * SS2_OUTPUT_SIZE * SS2_OUTPUT_SIZE, hipMemcpyDeviceToHost);
        verifySS2(verification, SS2_POST_ACT); //1.0f
		free(verification);
	}

	#endif 

#if LENET5_FC 
	// Performing Fully-Connected Computation
	kernel_fc1_lenet5_composite<<<(N33+K33-1)/K33, K33>>>((float(*)[SS2_OUTPUT_SIZE][SS2_OUTPUT_SIZE])ss2_layer.output,
									(float(*))fc_layer.pre_output, 
									(float(*)[CHANNEL2][SS2_OUTPUT_SIZE][SS2_OUTPUT_SIZE])fc_layer.weight,
									fc_layer.bias, (float(*))fc_layer.output);

	// Verifying Fully-Connected Computation
	if (verify) {
		printf("Veri Fully-Connected Lenet: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.pre_output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
        verifyFCfil(verification, SS2_POST_ACT*WEIGHT*CHANNEL2*SS2_OUTPUT_SIZE*SS2_OUTPUT_SIZE + BIAS);//216.0f
		
		printf("Veri Fully-Connected Lenet activation: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, FC_POST_ACT);//1.0f

		free(verification);
	}

#else 
	#if FC_COMPOSITE
	// Performing Fully-Connected Computation
	kernel_fc1_composite<<<(N3+K3-1)/K3, K3>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output,
	#else
	kernel_fc1<<<(N3+K3-1)/K3, K3>>>((float(*)[SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])ss_layer.output,
	#endif  
									(float(*))fc_layer.pre_output, 
									(float(*)[CHANNEL][SS_OUTPUT_SIZE][SS_OUTPUT_SIZE])fc_layer.weight,
									fc_layer.bias, (float(*))fc_layer.output);

	// Verifying Fully-Connected Computation
	if (verify) {
		printf("Veri Fully-Connected: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		#if FC_COMPOSITE
		hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, FC_POST_ACT);//1.0f
		#else
		hipMemcpy(verification, fc_layer.pre_output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
        verifyFCfil(verification, SS_POST_ACT*WEIGHT*CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE);//216.0f
		#endif 
		free(verification);
	}

	#if !FC_COMPOSITE	
	// Performing Fully-Connected bias operation
	kernel_fc1_bias<<<1, K3>>>((float(*))fc_layer.pre_output, fc_layer.bias);

	// Verifying Fully-Connected bias operation
	if (verify) {
		printf("Veri Fully-Connected bias: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.pre_output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFCfil(verification, SS_POST_ACT*WEIGHT*CHANNEL*SS_OUTPUT_SIZE*SS_OUTPUT_SIZE + BIAS); //217.0f
		free(verification);
	}

	// Performing Fully-Connected sigmoid operation
	kernel_fc1_sigmoid<<<1, K3>>>((float(*))fc_layer.pre_output, (float(*))fc_layer.output);

	// Verifying Fully-Connected sigmoid operation
	if (verify) {
		printf("Veri Fully-Connected sigmoid: ");
		verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
		hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
		verifyFC(verification, FC_POST_ACT); // 1.0f
		free(verification);
	}
	#endif // !FC_COMPOSITE

#endif //LENET5_FC
#endif //ALL_LAYERS 
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);


  #if 0
  if (print_status == 1 || print_status == 0 || print_status == -1 || print_status == -2 || print_status == -3 || print_status == -4){
    verification = (float*)malloc(sizeof(float) * NUM_CLASSES);
    hipMemcpy(verification, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);
  
      for(int i=0; i<NUM_CLASSES; i++){
        printf("%3.1f[%d] ", verification[i], i);
    }

    print_status--; 
    
    printf("\n");
    free(verification);
  }
  #endif 

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_input);
	//printf("\n"); 
	return elapsedTime;
}

void copy_trained_parameters() {
	hipMemcpy(conv_layer.weight, c1_weight, sizeof(float) * CHANNEL * FILTER_SIZE * FILTER_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(conv_layer.bias, c1_bias, sizeof(float) * CHANNEL, hipMemcpyHostToDevice);
	hipMemcpy(ss_layer.weight, s2_weight, sizeof(float) * SS_SIZE * SS_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(ss_layer.bias, s2_bias, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(fc_layer.weight, f3_weight, sizeof(float) * NUM_CLASSES * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE * SS_OUTPUT_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(fc_layer.bias, f3_bias, sizeof(float) * NUM_CLASSES, hipMemcpyHostToDevice);
}

int main() {
	int ret, i;
	mnist_data *test_set;
	static unsigned int test_cnt;

	// Calling the mnist_load() function
	if (ret = mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte", &test_set, &test_cnt) != 0) {
		printf("An error occured: %d\n", ret);
	} else {
		printf("test_cnt = %d\n", test_cnt);
	}

	// Verifying the image and label data of the specified number of examples
	//printExamples(&test_set, 1);

	// Verifying the convolutional layer
	double data[INSIZE][INSIZE];

 	//data[0][0] = 0.0f; 
	for (i = 0; i < INSIZE; i++) {
		for (int j = 0; j < INSIZE; j++){
			data[i][j] = INPUT; //1.0f;
      		//data[i][j+1] =  data[i][j] + 1.0f; 
    	}
	}	

	forward_pass(data, true);
	#if !SIMULATION
	copy_trained_parameters();
	#endif 

	// Performing forward pass
	unsigned int error = 0;
	unsigned int max = 0;
	float res[10];

	#if SIMULATION	
	for (i = 0; i < 1; i++) { //test_cnt
		time_taken += forward_pass(test_set[i].data, true);
	#else
	for (i = 0; i < test_cnt; i++) { //test_cnt
		time_taken += forward_pass(test_set[i].data, false);
	#endif 
		hipMemcpy(res, fc_layer.output, sizeof(float) * NUM_CLASSES, hipMemcpyDeviceToHost);

		for (int j = 0; j < NUM_CLASSES; j++) {
			if (res[max] < res[j])
				max = j;
		}

		if (max != test_set[i].label)
			error++;
	}

	printf("Error Rate = %f%% (%d out of 10000)\n", double(error) / double(test_cnt) * 100.0, error);
	printf("Accuracy = %.3f%% (%d out of 10000)\n", 100.0 - double(error) / double(test_cnt) * 100.0, test_cnt - error);
	printf("Execution time = %f (ms) \n\n", time_taken);

	free(test_set);
	return 0;
}
